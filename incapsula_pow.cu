#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdint>
#include <cstring>
#include <climits>
#include <chrono>
#include <iostream>
#include <hip/hip_runtime.h>

#define SHA1_BLOCK_SIZE 20

// Constant memory declarations
__constant__ uint8_t d_salt_const[16]; // 16-byte salt for PoW
__constant__ uint32_t d_target_const[5]; // 5x32-bit words = 160-bit target hash

// Rotate left utility
// Performs bitwise left rotation on a 32-bit word
__device__ __forceinline__ uint32_t rotate_left(uint32_t value, uint32_t count) {
    return (value << count) | (value >> (32 - count));
}

// Single-block SHA-1 transform
__device__ void sha1_transform(const uint8_t* data, uint32_t* state) {
    uint32_t w[80];
    // Pepare message schedule (first 16 words from data, big-endian)
	#pragma unroll
    for (int i = 0; i < 16; ++i) {
        w[i] = (uint32_t(data[i * 4]) << 24) |
            (uint32_t(data[i * 4 + 1]) << 16) |
            (uint32_t(data[i * 4 + 2]) << 8) |
            (uint32_t(data[i * 4 + 3]));
    }
	#pragma unroll
    // Extend to 80 words
    for (int i = 16; i < 80; ++i) {
        w[i] = rotate_left(w[i - 3] ^ w[i - 8] ^ w[i - 14] ^ w[i - 16], 1);
    }

    // Init working variables
    uint32_t a = state[0], b = state[1], c = state[2], d = state[3], e = state[4];

    // Main loop - 80 rounds
	#pragma unroll
    for (int i = 0; i < 80; ++i) {
        uint32_t f, k;
        if (i < 20) { f = (b & c) | (~b & d); k = 0x5A827999; }
        else if (i < 40) { f = b ^ c ^ d; k = 0x6ED9EBA1; }
        else if (i < 60) { f = (b & c) | (b & d) | (c & d); k = 0x8F1BBCDC; }
        else { f = b ^ c ^ d; k = 0xCA62C1D6; }
        uint32_t temp = rotate_left(a, 5) + f + e + k + w[i];
        e = d;
        d = c;
        c = rotate_left(b, 30);
        b = a;
        a = temp;
    }
    state[0] += a;
    state[1] += b;
    state[2] += c;
    state[3] += d;
    state[4] += e;
}

//-----------------------------------------------------------------------------------
// CUDA kernel: pow_kernel
// Each thread tests one nonce for a SHA-1 match to the target.
// 'startNonce' + thread-index = current nonce
// 'batchSize' limits how many total nonces per kernel launch
// 'result' is a device pointer to int: stores the smallest matching nonce via atomicMin
//-----------------------------------------------------------------------------------
__global__ void pow_kernel(uint32_t startNonce, uint32_t batchSize, int* result) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batchSize || *result != INT_MAX) return;

    uint32_t nonce = startNonce + idx;
    uint8_t message[64];
    // Build the 24-byte message: salt (16) + nonce (4 little-endian) + 4 zero bytes
	#pragma unroll
    for (int i = 0; i < 16; ++i) message[i] = d_salt_const[i];
    message[16] = uint8_t(nonce & 0xFF);
    message[17] = uint8_t((nonce >> 8) & 0xFF);
    message[18] = uint8_t((nonce >> 16) & 0xFF);
    message[19] = uint8_t((nonce >> 24) & 0xFF);

	#pragma unroll
    for (int i = 20; i < 24; ++i) {
        message[i] = 0;
    }

    // Standard SHA-1 padding: 0x80 then zeros until 56, then 64-bit length
    message[24] = 0x80;
	#pragma unroll
    for (int i = 25; i < 56; ++i) {
        message[i] = 0;
    }
    uint64_t bitLen = (uint64_t)24 * 8;

	#pragma unroll
    for (int i = 0; i < 8; ++i) message[56 + i] = uint8_t((bitLen >> ((7 - i) * 8)) & 0xFF);

    // Compute SHA-1
    uint32_t state[5] = { 0x67452301, 0xEFCDAB89, 0x98BADCFE, 0x10325476, 0xC3D2E1F0 };
    sha1_transform(message, state);

    // Compare to constant target
    bool match = true;
	#pragma unroll
    for (int i = 0; i < 5; ++i) {
        if (state[i] != d_target_const[i]) { match = false; break; }
    }

    if (match) {
        // atomically record smallest nonce that matches
        atomicMin(result, (int)nonce);
    }
}

int hexCharToInt(char c) {
    if (c >= '0' && c <= '9') return c - '0';
    if (c >= 'a' && c <= 'f') return c - 'a' + 10;
    if (c >= 'A' && c <= 'F') return c - 'A' + 10;
    return 0;
}


void hexToBytes(const char* hex, uint8_t* out, size_t len) {
    for (size_t i = 0; i < len; ++i) {
        out[i] = (hexCharToInt(hex[2 * i]) << 4) |
            hexCharToInt(hex[2 * i + 1]);
    }
}

int main(int argc, char** argv) {
    if (argc != 4) {
        std::printf("Usage: %s <T_ms> <D_hex20> <S_hex16>\n", argv[0]);
        return 1;
    }
    int T = std::atoi(argv[1]);     // time limit in milliseconds
    const char* Dhex = argv[2];     // target hash in hex (20 bytes)
    const char* Shex = argv[3];     // salt in hex (16 bytes)

    uint8_t h_salt[16]; hexToBytes(Shex, h_salt, 16);
    uint8_t h_targetBytes[20]; hexToBytes(Dhex, h_targetBytes, 20);
    uint32_t h_target[5];

    for (int i = 0; i < 5; ++i) {
        h_target[i] = (uint32_t(h_targetBytes[4 * i]) << 24) |
            (uint32_t(h_targetBytes[4 * i + 1]) << 16) |
            (uint32_t(h_targetBytes[4 * i + 2]) << 8) |
            (uint32_t(h_targetBytes[4 * i + 3]));
    }


    // Copy salt & target to GPU constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_salt_const), h_salt, 16);
    hipMemcpyToSymbol(HIP_SYMBOL(d_target_const), h_target, 5 * sizeof(uint32_t));

    int* d_result; hipMalloc(&d_result, sizeof(int));
    int h_result = INT_MAX;
    hipMemcpy(d_result, &h_result, sizeof(int), hipMemcpyHostToDevice);

    // Define kernel launch parameters
    const uint32_t batchSize = 1 << 20;
    const uint32_t threadsPerBlock = 256;
    uint32_t blocks = (batchSize + threadsPerBlock - 1) / threadsPerBlock;

    uint32_t startNonce = 0;
    uint64_t totalIterations = 0;

    // Start timing and loop until nonce found or time expires
    auto t_start = std::chrono::high_resolution_clock::now();
    auto t_limit = t_start + std::chrono::milliseconds(T);
    int foundNonce = -1;

    while (std::chrono::high_resolution_clock::now() < t_limit) {
        h_result = INT_MAX;
        hipMemcpy(d_result, &h_result, sizeof(int), hipMemcpyHostToDevice);

        // Launch kernel batch
        pow_kernel << <blocks, threadsPerBlock >> > (startNonce, batchSize, d_result);
        hipDeviceSynchronize();
        hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);
        if (h_result != INT_MAX) {
            foundNonce = h_result;
            totalIterations += uint64_t(foundNonce - startNonce) + 1;
            break;
        }

        // No match in this batch, continue
        totalIterations += batchSize;
        startNonce += batchSize;
    }

    // Timing end
    auto t_end = std::chrono::high_resolution_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(t_end - t_start).count();

    if (foundNonce >= 0) {
        std::printf("foundNonce: %d\niterations: %llu\ntimeTaken: %lld ms\n", foundNonce, totalIterations, (long long)elapsed);
    }
    else {
        std::printf("No valid nonce found in %llu iterations within %d ms\n", totalIterations, T);
    }

    hipFree(d_result);
    return 0;
}
